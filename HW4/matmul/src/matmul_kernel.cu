#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <chrono>
#include <assert.h>
#include "matmul.h"
using namespace std;


void allocateDeviceMemory(void **M, int size)
{
  hipError_t err = hipMalloc(M, size);
  assert(err == hipSuccess);
}

void deallocateDeviceMemory(void *M)
{
  hipError_t err = hipFree(M);
  assert(err == hipSuccess);
}

void matmul_ref(const int *const matrixA, const int *const matrixB,
                int *const matrixC, const int n)
{
  // You can assume matrixC is initialized with zero
  for (int i = 0; i < n; i++)
    for (int j = 0; j < n; j++)
      for (int k = 0; k < n; k++)
        matrixC[i * n + j] += matrixA[i * n + k] * matrixB[k * n + j];
}

__global__ void MatrixMulKernel(const int* d_A, const int* d_B, int* d_C, const int n);

void matmul_optimized(const int *const matrixA, const int *const matrixB,
                      int *const matrixC, const int *d_A, const int *d_B, int *const d_C, const int n)
{

  // TODO: Implement your CUDA code
  hipMemcpy((int *)d_A, (const int*)matrixA, n*n*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy((int *)d_B, (const int*)matrixB, n*n*sizeof(int), hipMemcpyHostToDevice);

  const int TILE_WIDTH = 32;
  dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
  dim3 dimGrid(n / TILE_WIDTH, n / TILE_WIDTH, 1);

  MatrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, n);
  hipMemcpy((int *)matrixC, (const int*)d_C, n*n*sizeof(int), hipMemcpyDeviceToHost);
}

__global__ void MatrixMulKernel(const int* d_A, const int* d_B, int* d_C, const int n)
{
  /**
    * Trial 1: Naive version 
    */
  // int row = blockIdx.y * blockDim.y + threadIdx.y;
  // int col = blockIdx.x * blockDim.x + threadIdx.x;

  // float temp = 0;
  // for (int k = 0; k < n; ++k){
  //   d_C[row * n + col] += d_A[row * n + k] * d_B[k * n + col];
  // }

  
  /**
    * Trial 2: Local variables on register
    */
  // int row = blockIdx.y * blockDim.y + threadIdx.y;
  // int col = blockIdx.x * blockDim.x + threadIdx.x;

  // float temp = 0;
  // for (int k = 0; k < n; ++k){
  //   temp += d_A[row * n + k] * d_B[k * n + col];
  // }
  // d_C[row * n + col] = temp;

  /**
    * Trial 3: Blocked matmul with shared memory
    */
  // const int TILE_WIDTH = 32;
  // __shared__ int subTileM[TILE_WIDTH][TILE_WIDTH];
  // __shared__ int subTileN[TILE_WIDTH][TILE_WIDTH];

  // int bx = blockIdx.x; int by = blockIdx.y;
  // int tx = threadIdx.x; int ty = threadIdx.y;

  // int Row = by * TILE_WIDTH + ty;
  // int Col = bx * TILE_WIDTH + tx;
  // float value = 0;

  // for(int m = 0; m < n/TILE_WIDTH; ++m){
  //   subTileM[ty][tx] = d_A[Row*n + m*TILE_WIDTH + tx];
  //   subTileN[ty][tx] = d_B[(m*TILE_WIDTH+ty)*n + Col];
  //   __syncthreads();

  //   for(int k = 0; k < TILE_WIDTH; ++k)
  //     value += subTileM[ty][k] * subTileN[k][tx];
  //   __syncthreads();
  // }
  // d_C[Row*n + Col] = value;
  /**
    * Trial 4: Boundary handling with Blocked matmul
    */
  const int TILE_WIDTH = 32;
  __shared__ int subTileM[TILE_WIDTH][TILE_WIDTH];
  __shared__ int subTileN[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x; int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  float value = 0;

  for(int m = 0; m < (n - 1) / TILE_WIDTH + 1; ++m){
    if (Row < n && m * TILE_WIDTH + tx < n)
      subTileM[ty][tx] = d_A[Row * n + m * TILE_WIDTH + tx];
    else 
      subTileM[ty][tx] = 0;

    if (Col < n && m * TILE_WIDTH + ty < n)
      subTileN[ty][tx] = d_B[(m * TILE_WIDTH + ty) * n + Col];
    else 
      subTileM[ty][tx] = 0;
    __syncthreads();

    for(int k = 0; k < TILE_WIDTH; ++k)
      value += subTileM[ty][k] * subTileN[k][tx];
    __syncthreads();
  }
  if (Row < n && Col < n)
    d_C[Row * n + Col] = value;  
}
